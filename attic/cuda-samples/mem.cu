
#include <hip/hip_runtime.h>
#include <stdio.h>


#define imin(a,b) (a<b?a:b)

const int N = 33 * 400 * 400;
const int threadsPerBlock = 1024;
const int blocksPerGrid =
            imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );


__global__ void dot( int size, double *a, double *b, double *c ) {
    __shared__ double cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    double   temp = 0;
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


double malloc_test( int size ) {
    hipEvent_t     start, stop;
    double           *a, *b, c, *partial_c;
    double           *dev_a, *dev_b, *dev_partial_c;
    float           elapsedTime;

    hipEventCreate( &start );
    hipEventCreate( &stop ) ;

    // allocate memory on the CPU side
    a = (double*)malloc( size*sizeof(double) );
    b = (double*)malloc( size*sizeof(double) );
    partial_c = (double*)malloc( blocksPerGrid*sizeof(double) );

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a,
                size*sizeof(double) ) ;
    hipMalloc( (void**)&dev_b,
                size*sizeof(double) ) ;
    hipMalloc( (void**)&dev_partial_c,
                              blocksPerGrid*sizeof(double) );

    // fill in the host memory with data
    for (int i=0; i<size; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    hipEventRecord( start, 0 ) ;
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, size*sizeof(double),
                hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size*sizeof(double),
                              hipMemcpyHostToDevice ) ;

    dot<<<blocksPerGrid,threadsPerBlock>>>( size, dev_a, dev_b,
                                            dev_partial_c );
    // copy the array 'c' back from the GPU to the CPU
     hipMemcpy( partial_c, dev_partial_c,
                 blocksPerGrid*sizeof(double),
                 hipMemcpyDeviceToHost ) ;

     hipEventRecord( stop, 0 ) ;
     hipEventSynchronize( stop ) ;
     hipEventElapsedTime( &elapsedTime,
                                        start, stop ) ;

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }

     hipFree( dev_a ) ;
     hipFree( dev_b ) ;
     hipFree( dev_partial_c ) ;

    // free memory on the CPU side
    free( a );
    free( b );
    free( partial_c );

    // free events
     hipEventDestroy( start  );
     hipEventDestroy( stop  );

    printf( "Value calculated:  %f\n", c );

    return elapsedTime;
}


double cuda_host_alloc_test( int size ) {
    hipEvent_t     start, stop;
    double           *a, *b, c, *partial_c;
    double           *dev_a, *dev_b, *dev_partial_c;
    float           elapsedTime;

    hipEventCreate( &start ) ;
    hipEventCreate( &stop ) ;

    // allocate the memory on the CPU
     hipHostAlloc( (void**)&a,
                 size*sizeof(double),
                 hipHostMallocWriteCombined |
                        hipHostMallocMapped ) ;
     hipHostAlloc( (void**)&b,
                 size*sizeof(double),
                 hipHostMallocWriteCombined |
                        hipHostMallocMapped ) ;
     hipHostAlloc( (void**)&partial_c,
                 blocksPerGrid*sizeof(double),
                 hipHostMallocMapped ) ;

    // find out the GPU pointers
     hipHostGetDevicePointer( &dev_a, a, 0  );
     hipHostGetDevicePointer( &dev_b, b, 0  );
     hipHostGetDevicePointer( &dev_partial_c,
                                            partial_c, 0  );

    // fill in the host memory with data
    for (int i=0; i<size; i++) {
        a[i] = i;
        b[i] = i*2;
    }

     hipEventRecord( start, 0 ) ;

    dot<<<blocksPerGrid,threadsPerBlock>>>( size, dev_a, dev_b,
                                            dev_partial_c );

  	hipDeviceSynchronize();
    hipEventRecord( stop, 0 ) ;
    hipEventSynchronize( stop ) ;
    hipEventElapsedTime( &elapsedTime,
                                        start, stop ) ;

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }

     hipHostFree( a ) ;
     hipHostFree( b ) ;
     hipHostFree( partial_c ) ;

    // free events
     hipEventDestroy( start ) ;
     hipEventDestroy( stop ) ;

    printf( "Value calculated:  %f\n", c );

    return elapsedTime;
}



// Main cuda function

int main() {
    printf("Hello from cuda \n");
    hipDeviceProp_t  prop;
    int whichDevice;
     hipGetDevice( &whichDevice ) ;
     hipGetDeviceProperties( &prop, whichDevice ) ;
    if (prop.canMapHostMemory != 1) {
        printf( "Device can not map memory.\n" );
        return 0;
    }

    double           elapsedTime;

    hipSetDeviceFlags( hipDeviceMapHost );

    // try it with malloc
    elapsedTime = malloc_test( N );
    printf( "Time using cudaMalloc:  %3.1f ms\n",
            elapsedTime );

    // now try it with cudaHostAlloc
    elapsedTime = cuda_host_alloc_test( N );
    printf( "Time using cudaHostAlloc:  %3.1f ms\n",
            elapsedTime );
    return 0;
}
