
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(double *x){
    x[threadIdx.x] = 2;
}

int main(){

	double *x;
  hipMallocManaged(&x, sizeof(double)*2);
  hipError_t error = hipGetLastError();
  printf("%s\n", error);
  x[0] = 0;
  x[1] = 0;

  kernel<<<1, 2>>>(x);
  hipDeviceSynchronize();

  printf("result = %f\n", x[0]);
  printf("result = %f\n", x[1]);

  hipFree(x);
  return 0;
}
